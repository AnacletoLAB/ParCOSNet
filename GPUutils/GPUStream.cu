#ifdef WIN32
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#endif

#include "GPUStream.h"

GPUStream::GPUStream( int n ) : numThreads( n ) {

    streams = new hipStream_t[numThreads];

    for (int i = 0; i < numThreads; i++)
        hipStreamCreate(&streams[i]);
		//hipStreamCreateWithFlags( &streams[i], hipStreamNonBlocking	);
}

GPUStream::~GPUStream() {

    for (int i = 0; i < numThreads; i++)
        hipStreamDestroy(streams[i]);

    delete[] streams;
}
