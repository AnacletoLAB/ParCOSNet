// COSnet - Cuda Cleaner class
// Alessandro Petrini, 2017
#include <iostream>
#include "GPUCudaCleaner.h"

CudaCleaner::CudaCleaner() {}

CudaCleaner::~CudaCleaner() {
	std::cout << "calling hipDeviceReset()..." << std::endl;
	hipDeviceReset();
	std::cout << "Done." << std::endl;
}
