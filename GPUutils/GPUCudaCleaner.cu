#include <iostream>
#include "GPUCudaCleaner.h"

CudaCleaner::CudaCleaner() {}

CudaCleaner::~CudaCleaner() {
	std::cout << "calling hipDeviceReset()..." << std::endl;
	hipDeviceReset();
	std::cout << "Done." << std::endl;
}
