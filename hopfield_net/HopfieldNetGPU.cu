#include "hip/hip_runtime.h"
// COSnet - Hopfield Net GPU class
// Alessandro Petrini, 2017
#ifdef WIN32
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#endif

#include <iostream>
#include <memory>
#include <algorithm>
#include <numeric>
#include <iomanip>
#include <math.h>
#include "hopfield_net/HopfieldNet.h"
#include "hopfield_net/HopfieldNetUtils.h"
#include "graph/graph.h"

#define ITERATION_LIMIT 5000

template<typename nodeW, typename edgeW>
HopfieldNetGPU<nodeW, edgeW>::HopfieldNetGPU( const Graph<nodeW, edgeW> * const inGraph_d, const Coloring * const inCol_d,
		float inPosState, float inNegState, float inRegulWeight ):
		HopfieldNet<nodeW, edgeW>( inGraph_d, inCol_d, inPosState, inNegState, inRegulWeight ),
		graph_d( inGraph_d ),
		col_d( inCol_d ) {

	hipError_t cuSts;
	this->hState.state = new float[this->hState.size];
	this->hState.score = new float[this->hState.size];

	hState_d.size = this->hState.size;
	cuSts = hipMalloc( (void**)&(hState_d.state), hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMalloc( (void**)&(hState_d.score), hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );

	numThreads = 32;
	threadsPerBlock = dim3( numThreads, 1, 1 );
}


template<typename nodeW, typename edgeW>
HopfieldNetGPU<nodeW, edgeW>::~HopfieldNetGPU() {
	hipError_t cuSts;
	cuSts = hipFree( hState_d.score ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipFree( hState_d.state ); cudaCheck( cuSts, __FILE__, __LINE__ );
	delete[] this->hState.score;
	delete[] this->hState.state;
}



///////////////////////////////////

template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::run_nodewise() {
	hipError_t cuSts;

#ifdef PRINTHOPFIELDTITLE
	std::cout << "\033[32;1m** Hopfiled Net GPU alternative runner **\033[0m" << std::endl;
#endif

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	this->numIter = 0;
	bool modified = true;
	bool *modified_d;
	cuSts = hipMalloc( (void**) &modified_d, sizeof(bool) ); cudaCheck(cuSts,__FILE__,__LINE__);

	std::unique_ptr<uint32_t[]> ISsize_h( new uint32_t[col_d->nCol + 1] );
	cuSts = hipMemcpy( ISsize_h.get(), col_d->cumulSize, (col_d->nCol + 1) * sizeof( uint32_t ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );

#ifdef VERBOSEHOPFIELD
	printf( "Numero colori: %d\n", col->nCol );
	for ( int i = 0; i < col->nCol; i++)
		printf( "colore %d: %d\n", i, ISsize_h[i] );
#endif

	hipEventRecord( start );
	while (modified) {

		this->numIter++;
		cuSts = hipMemset( modified_d, false, sizeof(bool) ); cudaCheck( cuSts, __FILE__, __LINE__ );

		for (uint32_t ISidx = 0; ISidx < col_d->nCol; ISidx++) {

			uint32_t numberOfNodes = ISsize_h[ISidx + 1] - ISsize_h[ISidx];
			blocksPerGrid = dim3( (numberOfNodes + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1 );

			// launch the Hopfield kernel
			HopfieldNetGPU_k::updateIS_nodewise<nodeW, edgeW> <<<blocksPerGrid, threadsPerBlock >>> (
					hState_d.state,
					hState_d.score,
					graph_d->getStruct()->cumulDegs, graph_d->getStruct()->edgeWeights, graph_d->getStruct()->neighs, graph_d->getStruct()->nodeThresholds,
					graph_d->getStruct()->nNodes,
					col_d->nCol, col_d->colClass, col_d->cumulSize,
					ISidx,
					modified_d,
					this->posState,
					this->negState,
					this->regulWeight
					);

			hipDeviceSynchronize();
			cuSts = hipGetLastError(); cudaCheck( cuSts, __FILE__, __LINE__ );

		}

		cuSts = hipMemcpy(&modified, modified_d, sizeof(bool), hipMemcpyDeviceToHost); cudaCheck( cuSts, __FILE__, __LINE__ );
		if (this->numIter > ITERATION_LIMIT) {
			std::cout << "Massimo numero di iterazioni raggiunto!!! Uscita forzata" << std::endl;
			break;
		}
	}

	cuSts = hipEventRecord(stop); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipEventSynchronize(stop); cudaCheck( cuSts, __FILE__, __LINE__ );
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
#ifdef VERBOSEHOPFIELD
	std::cout << "Stabilita' raggiunta in " << numIter << " iterazioni" << std::endl;
#endif

	// final state & log
	cuSts = hipMemcpy( this->hState.state, this->hState_d.state, this->hState.size * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );
	//HL->GPUrunTime = milliseconds / 1000;
	//HL->GPUnumIter = num_iter;
	//HL->speedup = HL->runTime / HL->GPUrunTime;
	cuSts = hipFree(modified_d); cudaCheck( cuSts, __FILE__, __LINE__ );

	cuSts = hipEventDestroy( stop ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipEventDestroy( start ); cudaCheck( cuSts, __FILE__, __LINE__ );
}

template<typename nodeW, typename edgeW>
__global__ void HopfieldNetGPU_k::updateIS_nodewise
			( float * const state, float * const score,
			node_sz * cumulDegs, edgeW * edgeWeights, node * neighs_, nodeW * nodeThresholds,
			const node_sz nNodes,
			const uint32_t nCol, const uint32_t	* const colClass, const uint32_t * const cumulSize,
			const int colorIdx,
			bool * const modified_d,
			const float posState, const float negState, const float regulWeight ) {

	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid >= (cumulSize[colorIdx + 1] - cumulSize[colorIdx]))
		return;

	float newScore = 0;

	const int 			offsetCol = cumulSize[colorIdx];						// offset per il coloring
	const int 			node   = colClass[offsetCol + tid];
	const int			offset = cumulDegs[node];
	const int 			degree = cumulDegs[node + 1] - offset;
	const edgeW * const weights = &(edgeWeights[offset]);
	const uint32_t	* const neighs  = &(neighs_[offset]);
	unitVal oldState = state[node];

	for (int i = 0; i < degree; i++) {
		newScore += (weights[i] - regulWeight) * state[neighs[i]];
	}
	__syncthreads(); // Non dovrebbe servire

	// modifica per regolarizzazione
	uint32_t nodoreg;
	for (uint32_t i = 0; i < nCol; i++) {
		if (i == colorIdx)
			continue;
		else {
			uint32_t IS_size = cumulSize[i + 1] - cumulSize[i];
			for (uint32_t k = 0; k < IS_size; k++) {
				nodoreg = colClass[cumulSize[i] + k];
				newScore -= state[nodoreg] * regulWeight;
			}
		}
	}
	__syncthreads();

	score[node] = newScore - nodeThresholds[node];
	state[node] = SIGNTH( (newScore - nodeThresholds[node]) );

	if (state[node] != oldState) {
		*modified_d = true;
	}
}


////////////////////////////////////////

template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::run_edgewise() {
	hipError_t cuSts;

#ifdef PRINTHOPFIELDTITLE
	std::cout << "\033[32;1m** Hopfiled Net GPU runner **\033[0m" << std::endl;
#endif

	//timer cudaEvent per Benchmark
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//conto iterazioni e criterio di arresto
	this->numIter = 0;
	bool modified = true;
	bool *modified_d;
	cuSts = hipMalloc( (void**) &modified_d, sizeof(bool) ); cudaCheck( cuSts, __FILE__, __LINE__ );

	//alloco e copio cumulSize
	std::unique_ptr<uint32_t[]> cumulSize_h( new uint32_t[ (col_d->nCol+1) ] );
	cuSts = hipMemcpy( cumulSize_h.get(), col_d->cumulSize, (col_d->nCol+1) * sizeof( uint32_t ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );

#ifdef VERBOSEHOPFIELD
	printf( "Numero colori: %d\n", col->nCol );
	for ( int i = 0; i < col->nCol; i++)
		printf( "colore %d: %d\n", i, ISsize_h[i] );
#endif

	// **  run net on device: loop on ISs  **
	CHECK( hipEventRecord( start ) );
	while ( modified ) {

		this->numIter++;
		cuSts = hipMemset( modified_d, false, sizeof(bool) ); cudaCheck( cuSts, __FILE__, __LINE__ );

		// update all ISs
		for (uint32_t ISidx = 0; ISidx < col_d->nCol; ISidx++) {
			// col.meanUnitDeg non implementato nel colorer.
			/*
			int numThreads = pow(2, floor(log(col.meanUnitDeg[ISidx]) / log(2)));
			if (numThreads < 32)
				numThreads = 32;
			if (numThreads > 1024)
				numThreads = 1024;
			*/
			//int numThreads = 32;
			//dim3 blocksize(numThreads);			// num threads = average deg IS nodes crop to (32,1024)
			// col.ISsize[] inaccessibile da host.
			//dim3 gridsize(col.ISsize[ISidx]);	// num blocks = IS size
			//dim3 gridsize( ISsize_h[ISidx] );

			uint32_t colorSize = cumulSize_h[ISidx + 1] - cumulSize_h[ISidx];
			blocksPerGrid = dim3( colorSize, 1, 1 );

			// launch the Hopfield kernel
			HopfieldNetGPU_k::updateIS_edgewise<<<blocksPerGrid, threadsPerBlock, numThreads * sizeof(float)>>>(
					hState_d.state,
					hState_d.score,
					graph_d->getStruct()->cumulDegs, graph_d->getStruct()->edgeWeights, graph_d->getStruct()->neighs, graph_d->getStruct()->nodeThresholds,
					graph_d->getStruct()->nNodes,
					col_d->nCol, col_d->colClass, col_d->cumulSize,
					ISidx,
					modified_d,
					this->posState,
					this->negState,
					this->regulWeight
				);

			hipDeviceSynchronize();
			cuSts = hipGetLastError(); cudaCheck( cuSts, __FILE__, __LINE__ );

		}

		cuSts = hipMemcpy(&modified, modified_d, sizeof(bool), hipMemcpyDeviceToHost); cudaCheck( cuSts, __FILE__, __LINE__ );
		if (this->numIter > ITERATION_LIMIT) {
			std::cout << "Massimo numero di iterazioni raggiunto!!! Uscita forzata" << std::endl;
			break;
		}
	}

	cuSts = hipEventRecord(stop); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipEventSynchronize(stop); cudaCheck( cuSts, __FILE__, __LINE__ );
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
#ifdef VERBOSEHOPFIELD
	std::cout << "Stabilita' raggiunta in " << numIter << " iterazioni" << std::endl;
#endif

	// final state & log
	cuSts = hipMemcpy(this->hState.state, hState_d.state, this->hState.size * sizeof( unitVal ), hipMemcpyDeviceToHost); cudaCheck( cuSts, __FILE__, __LINE__ );
	//HL->GPUrunTime = milliseconds / 1000;
	//HL->GPUnumIter = num_iter;
	//HL->speedup = HL->runTime / HL->GPUrunTime;
	cuSts = hipFree(modified_d); cudaCheck( cuSts, __FILE__, __LINE__ );

	cuSts = hipEventDestroy( stop ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipEventDestroy( start ); cudaCheck( cuSts, __FILE__, __LINE__ );
}


template<typename nodeW, typename edgeW>
__global__ void HopfieldNetGPU_k::updateIS_edgewise( float * const state, float * const score,
		node_sz * cumulDegs, edgeW * edgeWeights, node * neighs_, nodeW * nodeThresholds,
		const node_sz nNodes,
		const uint32_t nCol, const uint32_t	* const colClass, const uint32_t * const cumulSize,
		const int colorIdx,
		bool * const modified_d,
		const float posState, const float negState, const float regulWeight  ) {

	// ID del thread all'interno del nodo, serve per la parallel reduction sum
	unsigned int tid = threadIdx.x;
	// ID del blocco nella griglia, serve per il calcolo dello score edgewise
	unsigned int bid = blockIdx.x;
	// DIM del blocco 1D, serve per il calcolo dello score edgewise
	// serve per segmentare il calcolo/quanti edge associo ad un thread
	// nel caso il numero di vicini del nodo corrente superi blockDim
	unsigned int dim = blockDim.x;

	// color idx supera numero colori?
	if (colorIdx >= nCol)
		return;

	const int 		offsetCol = cumulSize[colorIdx];

	// il kernel non deve prendere nodi nodeIdx al di fuori del colore attuale
	if( (offsetCol + bid) >= cumulSize[colorIdx + 1] )
		return;

	const int 		nodeIdx = colClass[offsetCol + bid];

	extern __shared__ float smem[];

	const int 		offsetDeg	= cumulDegs[nodeIdx];
	const int 		degree = cumulDegs[nodeIdx+1] - offsetDeg;

	// Indica il numero di vicini che ogni thread deve cuccarsi
	// es. se numero vicini = 146 e numero thread per blocco = 32 =>
	// 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 4 4 4 4 4 4 4 4 4 4 4 4 4 4
	// es. se numero vicini = 6 e numero thread per blocco = 32 =>
	// 1 1 1 1 1 1 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0
	// verifica con:
	// 		int nn = 146; int tPerBlk = 32;
    //		for (int tid = 0; tid < tPerBlk; tid++)
    //		    std::cout << nn / tPerBlk + ((nn % tPerBlk) > tid) << " ";}
	int neighPerThread = degree / blockDim.x + ((degree % blockDim.x) > tid);

	smem[tid] = 0;
	__syncthreads();

	for (int i = 0; i < neighPerThread; i++) {
		int indx = neighs_[offsetDeg + i * dim + tid];
		smem[tid] += edgeWeights[offsetDeg + i * dim + tid] * state[indx];

#ifdef DEBUGPRINTK_IS
		if (bid == 0)
			printf("node: %d tid: %d nperthisThread: %d i: %d idx: %d smem[tid]: %f\n", nodeIdx, tid, neighPerThread, i, indx, smem[tid]);
#endif

	}
	__syncthreads();

	// sum cache by parallel reduction
	for (unsigned int i = dim / 2; i > 32; i >>= 1) {
		if (tid < i)
			smem[tid] += smem[tid + i];
		__syncthreads();
	}
	// last warp
	if (tid < 16) {
		smem[tid] += smem[tid + 16];
		__syncthreads();
		smem[tid] += smem[tid + 8];
		__syncthreads();
		smem[tid] += smem[tid + 4];
		__syncthreads();
		smem[tid] += smem[tid + 2];
		__syncthreads();
		smem[tid] += smem[tid + 1];
		__syncthreads();
	}
	//__syncthreads();

#ifdef DEBUGPRINTK_IS
	// Naive reduction usata in fase di test...
	if (tid == 0) {
		//for( int i = 1; i < dim; i++) {
		//	smem[0] += smem[i];
		//}
		printf("node: %d tid: %d smem[tid]: %f\n", nodeIdx, tid, smem[tid]);
	}
#endif

	// update state
	if (tid == 0) {
		unitVal oldState = state[nodeIdx];

		// modifica per regolarizzazione
		uint32_t nodoreg;
		for (uint32_t i = 0; i < nCol; i++) {
			if (i == colorIdx)
				continue;
			else {
				uint32_t IS_size = cumulSize[i + 1] - cumulSize[i];
				for (uint32_t k = 0; k < IS_size; k++) {
					nodoreg = colClass[cumulSize[i] + k];
					smem[0] -= state[nodoreg] * regulWeight;
				}
			}
		}

		// aggiorno state e score
		score[nodeIdx] = smem[0] - nodeThresholds[nodeIdx];
		state[nodeIdx] = SIGNTH((smem[0] - nodeThresholds[nodeIdx]));

		//controllo se lo stato è stato modificato
		if (state[nodeIdx] != oldState) {
			*modified_d = true;
		}
	}
}





//////////////////////////////////////////////////////////

// va lasciato hipMemset a 0 per hScore?
template<typename nodeW, typename edgeW>
	void HopfieldNetGPU<nodeW, edgeW>::setInitState( const unitVal * const inState, const  unitVal* const inScore ) {
		hipError_t cuSts;
		for (int i = 0; i < this->hState.size; i++){
			this->hState.state[i] = static_cast<float>(inState[i]);
			this->hState.score[i] = 0;
		}
		cuSts = hipMemcpy( hState_d.state, this->hState.state, hState_d.size * sizeof( unitVal ), hipMemcpyHostToDevice ); cudaCheck( cuSts, __FILE__, __LINE__ );
		cuSts = hipMemset( hState_d.score, 0, hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
	}

// setta tuti gli initial state = inValue
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::setInitState( const unitVal inValue ) {
	hipError_t cuSts;
	cuSts = hipMemset( hState_d.state, inValue, hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMemset( hState_d.score, 0, hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
}

// setta a 0 state e score su memoria device
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::clearInitState() {
	hipError_t cuSts;
	cuSts = hipMemset(hState_d.state, 0, hState_d.size * sizeof( unitVal )); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMemset(hState_d.score, 0, hState_d.size * sizeof( unitVal )); cudaCheck( cuSts, __FILE__, __LINE__ );
}

// GPURandomizer riempie casualmente state e score su memoria device
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::setRandomInitState( GPURand * const randomizer ) {
	//randomizer->fillRandom( hState_d.state, hState_d.size );
}

template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::setInitStateProb( Prob p, char type ) {
	hipError_t cuSts;
	if ( type == 'z') {
		std::fill( this->hState.state, this->hState.state + this->hState.size, (-0.5 < 0 ? this->negState : this->posState) );
	} else if ( type == 'o') {
		std::fill( this->hState.state, this->hState.state + this->hState.size, (0.5 < 0 ? this->negState : this->posState) );
	} else if ( type == 'r') {
		unitVal pS = this->posState;
		unitVal nS = this->negState;
		std::generate( this->hState.state, this->hState.state + this->hState.size, [p, pS, nS](){return SIGNTHLAMBDA(p-randf(0, 1));} );
	}
	cuSts = hipMemcpy(hState_d.state, this->hState.state, hState_d.size * sizeof( unitVal ), hipMemcpyHostToDevice ); cudaCheck( cuSts, __FILE__, __LINE__ );
}

// ritorna i valori di state e score
// serve perchè hState e hState_d sono campi protected
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::returnVal( float * const inState, float * const inScore ) {
	hipError_t cuSts;
	cuSts = hipMemcpy(this->hState.state, hState_d.state, hState_d.size * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMemcpy(this->hState.score, hState_d.score, hState_d.size * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );
	for (int i = 0; i < hState_d.size; i++) {
		inState[i] = this->hState.state[i];
		inScore[i] = this->hState.score[i];
	}
}

// Funzione di test per valutare correttezza del kernel "accumulateScores"
// template<typename nodeW, typename edgeW>
// void HopfieldNetGPU<nodeW, edgeW>::normalizeScore( const GraphStruct<nodeW, edgeW> * const bigGraph, const uint32_t *const reduxToFull, const edgeW * const sumOfWghs_h ) {
// 	hipError_t cuSts;
// 	uint32_t n = graph_d->getStruct()->nNodes;
// 	uint32_t nOrig = bigGraph->nNodes;
// 	dim3 threadPerBlk( TPB_ACCUMUL, 1, 1 );
// 	uint32_t bPg = (n + 2 * threadPerBlk.x - 1) / (2 * threadPerBlk.x);
// 	dim3 blocksPerGrd( bPg, 1, 1 );
//
// 	unitVal		*	input = new unitVal[n];
// 	unitVal		*	input_d;
// 	unitVal		*	accumulatedScores_h = new unitVal[bPg];
// 	unitVal		*	accumulatedScores_d;
//
// 	cuSts = hipMalloc( (void**)&input_d, n * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
// 	cuSts = hipMalloc( (void**)&accumulatedScores_d, bPg * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
// 	std::fill( input, input + n, 1.0 );
// 	cuSts = hipMemcpy( input_d, input, n * sizeof( unitVal ), hipMemcpyHostToDevice );
// 	HopfieldNetGPU_k::accumulateScores <<<blocksPerGrd, threadPerBlk>>> ( n, input_d, accumulatedScores_d );
// 	hipDeviceSynchronize();
// 	cuSts = hipGetLastError(); cudaCheck( cuSts, __FILE__, __LINE__ );
// 	cuSts = hipMemcpy( accumulatedScores_h, accumulatedScores_d, bPg * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );
// 	unitVal totScore_d = std::accumulate( accumulatedScores_h, accumulatedScores_h + bPg, 0.0 );
// 	unitVal totScore_h = std::accumulate( input, input + n, 0.0 );
//
//
// 	uint32_t bPg2 = (n + threadPerBlk.x - 1) / (threadPerBlk.x);
// 	unitVal		*	accumulatedScores2_h = new unitVal[bPg2];
// 	unitVal		*	accumulatedScores2_d;
// 	cuSts = hipMalloc( (void**)&accumulatedScores2_d, bPg2 * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
// 	dim3 blocksPerGrd2( bPg2, 1, 1 );
// 	HopfieldNetGPU_k::accumulateScores2 <<<blocksPerGrd2, threadPerBlk>>> ( n, input_d, accumulatedScores2_d );
// 	hipDeviceSynchronize();
// 	cuSts = hipGetLastError(); cudaCheck( cuSts, __FILE__, __LINE__ );
// 	cuSts = hipMemcpy( accumulatedScores2_h, accumulatedScores2_d, bPg2 * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );
// 	unitVal totScore2_d = std::accumulate( accumulatedScores2_h, accumulatedScores2_h + bPg2, 0.0 );
// 	std::cout << "n: " << n << " totScore_d: " << totScore_d << " totScore2_d: " << totScore2_d << " - totScore_h: " << totScore_h << std::endl;
// 	// if (totScore_d != totScore_h) {
// 	// 	std::cout << "totScore_d: " << totScore_d << " - totScore_h: " << totScore_h << std::endl;
// 	// 	abort();
// 	// }
// }



template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::normalizeScore( const GraphStruct<nodeW, edgeW> * const bigGraph, const uint32_t *const reduxToFull, const edgeW * const sumOfWghs_h ) {
	hipError_t	cuSts;
	uint32_t	n = graph_d->getStruct()->nNodes;
	uint32_t	nOrig = bigGraph->nNodes;
	dim3		threadPerBlk( TPB_ACCUMUL, 1, 1 );
	uint32_t	bPg = (n + 2 * threadPerBlk.x - 1) / (2 * threadPerBlk.x);
	dim3		blocksPerGrd( bPg, 1, 1 );

	unitVal		*	accumulatedScores;
	uint32_t	*	reduxToFull_d;
	unitVal		*	sumOfWghs_d;
	cuSts = hipMalloc( (void**)&accumulatedScores,	bPg * sizeof( unitVal ) );		cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMalloc( (void**)&reduxToFull_d,		n * sizeof( uint32_t ) );		cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMalloc( (void**)&sumOfWghs_d,		nOrig * sizeof( unitVal ) );	cudaCheck( cuSts, __FILE__, __LINE__ );
	std::unique_ptr<unitVal[]> 	accumulatedScores_h( new unitVal[bPg] );

	// Calcolo somma degli scores
	HopfieldNetGPU_k::accumulateScores <<<blocksPerGrd, threadPerBlk>>> ( n, hState_d.score, accumulatedScores );
	cuSts = hipGetLastError(); cudaCheck( cuSts, __FILE__, __LINE__ );

		// uint32_t bPg2 = (n + threadPerBlk.x - 1) / (threadPerBlk.x);
		// unitVal		*	accumulatedScores2_h = new unitVal[bPg2];
		// unitVal		*	accumulatedScores2_d;
		// cuSts = hipMalloc( (void**)&accumulatedScores2_d, bPg2 * sizeof( unitVal ) ); cudaCheck( cuSts, __FILE__, __LINE__ );
		// dim3 blocksPerGrd2( bPg2, 1, 1 );
		// HopfieldNetGPU_k::accumulateScores2 <<<blocksPerGrd2, threadPerBlk>>> ( n, hState_d.score, accumulatedScores2_d );
		// hipDeviceSynchronize();
		// cuSts = hipGetLastError(); cudaCheck( cuSts, __FILE__, __LINE__ );
		// cuSts = hipMemcpy( accumulatedScores2_h, accumulatedScores2_d, bPg2 * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );
		// unitVal totScore2_d = std::accumulate( accumulatedScores2_h, accumulatedScores2_h + bPg2, 0.0 );



	// accumulazione della somma dei pesi dei nodi unlabelled
	float accumulatedWDeg = 0.0;
	for (uint32_t j = 0; j < n; j++) {
		accumulatedWDeg += sumOfWghs_h[reduxToFull[j]];
	}
	hipDeviceSynchronize();

	cuSts = hipMemcpy( accumulatedScores_h.get(), accumulatedScores, bPg * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMemcpy( sumOfWghs_d, sumOfWghs_h, nOrig * sizeof( unitVal ), hipMemcpyHostToDevice ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipMemcpy( reduxToFull_d, reduxToFull, n * sizeof( uint32_t ), hipMemcpyHostToDevice ); cudaCheck( cuSts, __FILE__, __LINE__ );

	// finisco l'accumulazione degli score su CPU
	unitVal totScore = std::accumulate( accumulatedScores_h.get(), accumulatedScores_h.get() + bPg, 0.0 );

			// unitVal * temphStateScores = new unitVal[n];
			// cuSts = hipMemcpy( temphStateScores, hState_d.score, n * sizeof( unitVal ), hipMemcpyDeviceToHost );
			// unitVal tempAccScores = 0.0f;
			// std::for_each( temphStateScores, temphStateScores + n, [&tempAccScores]( unitVal nn ) {tempAccScores += fabs( nn ); } );
			// //if (totScore != tempAccScores) {
			// //	std::cout << "errore nella reduction sugli score. GPU = " << totScore << " - CPU: " << tempAccScores << std::endl;
			// //	abort();
			// //}
			// //std::cout << std::setprecision(6) << "n: " << n << " - Host - k1: " << tempAccScores - totScore << " - Host - k2: " << tempAccScores - totScore2_d << std::endl;
			// //std::cout << "n: " << n << " totScore: " << totScore << " totScore2_d: " << totScore2_d << " - host: " << tempAccScores << std::endl;
			// delete[] temphStateScores;
			// totScore = tempAccScores;

	bPg = (n + threadPerBlk.x - 1) / threadPerBlk.x;
	blocksPerGrd = dim3( bPg, 1, 1 );
	HopfieldNetGPU_k::normalizeScores <<<blocksPerGrd, threadPerBlk>>> ( n, accumulatedWDeg, totScore, sumOfWghs_d, reduxToFull_d, hState_d.score );
	hipDeviceSynchronize();
	cuSts = hipGetLastError(); cudaCheck( cuSts, __FILE__, __LINE__ );

	cuSts = hipFree( sumOfWghs_d ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipFree( reduxToFull_d ); cudaCheck( cuSts, __FILE__, __LINE__ );
	cuSts = hipFree( accumulatedScores ); cudaCheck( cuSts, __FILE__, __LINE__ );
}


// __global__ void HopfieldNetGPU_k::accumulateScores2( const uint32_t unlab, const unitVal * const scores, unitVal * const accumScores ) {
// 	uint32_t baseBlock = blockDim.x * blockIdx.x;
//
// 	accumScores[blockIdx.x] = 0.0f;
// 	if (threadIdx.x==0) {
// 		for(uint32_t i = 0; i < blockDim.x; i++) {
// 			if (baseBlock + i < unlab)
// 				accumScores[blockIdx.x] += fabsf( scores[baseBlock + i] );
// 		}
// 	}
// }

__global__ void HopfieldNetGPU_k::accumulateScores( const uint32_t unlab, const unitVal * const scores, unitVal * const accumScores ) {

	__shared__ float tempScores[TPB_ACCUMUL];

	uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= unlab / 2)
		return;
	uint32_t baseBlock = 2 * blockDim.x * blockIdx.x;
	tempScores[threadIdx.x] = fabsf(scores[baseBlock + threadIdx.x]);
	__syncthreads();

	uint32_t incremento = ((baseBlock + threadIdx.x + blockDim.x) < unlab) ? blockDim.x : (unlab % blockDim.x) / 2;
	tempScores[threadIdx.x] += fabsf(scores[baseBlock + threadIdx.x + incremento]);
	__syncthreads();

#pragma unroll
	for (uint32_t i = blockDim.x / 2; i > 0; i >>= 1) {
		if ((threadIdx.x < i) & (tid + i < unlab / 2)){
			tempScores[threadIdx.x] += tempScores[threadIdx.x + i];
		}
		__syncthreads();
	}

	// Mi prendo cura degli ultimi elementi dei vettori score e deg nel caso in cui
	// i nodi siano dispari
	if ((unlab % 2) && (tid == 0)) {
		tempScores[0] += fabsf(scores[unlab - 1]);
	}
	__syncthreads();

	accumScores[blockIdx.x] = tempScores[0];

	return;
}


__global__ void HopfieldNetGPU_k::normalizeScores( const uint32_t unlab, const float accumWDeg, const unitVal accumScores,
	const unitVal * const sumOfWeights, const uint32_t * const indexes, unitVal * const scores ) {

	uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= unlab)
		return;
	scores[tid] = sumOfWeights[indexes[tid]] / accumWDeg + scores[tid] / accumScores;

	return;
}



//// Questo serve per mantenere le dechiarazioni e definizioni in classi separate
//// E' necessario aggiungere ogni nuova dichiarazione per ogni nuova classe tipizzata usata nel main
//template class HopfieldNetGPU<col, col>;
template class HopfieldNetGPU<float, float>;
