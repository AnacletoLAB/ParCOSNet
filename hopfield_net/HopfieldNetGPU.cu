#include "hip/hip_runtime.h"
#ifdef WIN32
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#endif

#include <iostream>
#include <memory>
#include <algorithm>
#include <numeric>
#include <math.h>
#include "hopfield_net/HopfieldNet.h"
#include "hopfield_net/HopfieldNetUtils.h"
#include "graph/graph.h"

#define ITERATION_LIMIT 5000

/*
#	#	#	#	#	#	#	#	#	#	#	#	#

				HopfieldNetGPU CLASS

#	#	#	#	#	#	#	#	#	#	#	#	#
*/

/*
 *  CLASS HOPFIELDNETGPU: Hopfield Net run by GPU
 */
// Attenzione: si sta dando per scontato che il coloring passato al costruttore di
// HopfieldNetGPU sia un coloring_d già presente sulla GPU

 //COME FA A RIEMPIRE hState.size nel costruttore HopfieldNet se il grafo è in device?
template<typename nodeW, typename edgeW>
HopfieldNetGPU<nodeW, edgeW>::HopfieldNetGPU( const Graph<nodeW, edgeW> * const inGraph_d, const Coloring * const inCol_d,
		float inPosState, float inNegState, float inRegulWeight ):
		HopfieldNet<nodeW, edgeW>( inGraph_d, inCol_d, inPosState, inNegState, inRegulWeight ),
		graph_d( inGraph_d ),
		col_d( inCol_d ) {

			hipError_t cuSts;
			this->hState.state = new float[this->hState.size];
			this->hState.score = new float[this->hState.size];

			hState_d.size = this->hState.size;
			cuSts = hipMalloc( (void**)&(hState_d.state), hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts );
			cuSts = hipMalloc( (void**)&(hState_d.score), hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts );

			numThreads = 32;
			threadsPerBlock = dim3( numThreads, 1, 1 );
		};

/*
 *  Virttual destructor
 */
template<typename nodeW, typename edgeW>
HopfieldNetGPU<nodeW, edgeW>::~HopfieldNetGPU() {
	hipError_t cuSts;
	cuSts = hipFree( hState_d.score ); cudaCheck( cuSts );
	cuSts = hipFree( hState_d.state ); cudaCheck( cuSts );
	delete[] this->hState.score;
	delete[] this->hState.state;
}



/*
#	#	#	#	#	#	#	#	#	#	#	#	#

		HopfieldNetGPU run & updateIS NODEWISE

#	#	#	#	#	#	#	#	#	#	#	#	#
*/

//		MAIN MEMBER
//		SINGLE THREAD

/*
	NB ricorda, usa un coloring già in memoria device

	RUN FLOW:
		1 variabili per benchmark, iterazioni e criterio di arresto
		2 copio cumulsize del coloring
		3 CICLO WHILE (finchè gli stati della rete vengono modificati o ITERATION_LIMIT)
			4 CICLO FOR (sui colori/indipendent set)
				lancio dei kernel per ogni colore

*/

template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::run_nodewise() {
	hipError_t cuSts;

#ifdef PRINTHOPFIELDTITLE
	std::cout << "\033[32;1m** Hopfiled Net GPU alternative runner **\033[0m" << std::endl;
#endif

	//timer cudaEvent per Benchmark
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//conto iterazioni e criterio di arresto
	this->numIter = 0;
	bool modified = true;
	bool *modified_d;
	cuSts = hipMalloc( (void**) &modified_d, sizeof(bool) ); cudaCheck( cuSts );
	cuSts = hipMemset( modified_d, true, sizeof( bool ) ); cudaCheck( cuSts ); //forse non serve ma andiamo sul sicuro

	//alloco e copio cumulSize
	std::unique_ptr<uint32_t[]> cumulSize_h( new uint32_t[ (col_d->nCol+1) ] );
	cuSts = hipMemcpy( cumulSize_h.get(), col_d->cumulSize, (col_d->nCol+1) * sizeof( uint32_t ), hipMemcpyDeviceToHost ); cudaCheck( cuSts );

#ifdef VERBOSEHOPFIELD
	printf( "Numero colori: %d\n", col_d->nCol );
	for ( int i = 0; i < col_d->nCol; i++)
		printf( "colore %d: %d\n", i, cumulSize_h[i] );
#endif

	hipEventRecord( start );

	//ciclo finchè non si è stabilizzato numItert=ITERATION_LIMIT
	while (modified) {
		(this->numIter)++;
		cuSts = hipMemset( modified_d, false, sizeof(bool) ); cudaCheck( cuSts );

		//ciclo sui colori
		for (uint32_t ISidx = 0; ISidx < col_d->nCol; ISidx++) {
			uint32_t numberOfNodes = cumulSize_h[ISidx + 1] - cumulSize_h[ISidx];

			blocksPerGrid = dim3( (numberOfNodes + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1 );

			// launch the Hopfield kernel
			HopfieldNetGPU_k::updateIS_nodewise<<<blocksPerGrid, threadsPerBlock, numberOfNodes * sizeof(float)>>>(
					hState_d.state,			// net state
					hState_d.score,			// net score
					graph_d->getStruct(),	// graph structure (neighs, weighs, thresholds)
					col_d,					// graph coloring
					ISidx,					// indipendent set/color index
					modified_d,				// stop cond
					this->posState,
					this->negState
					);

			hipDeviceSynchronize();
			cuSts = hipGetLastError();
			cudaCheck2(cuSts,__FILE__,__LINE__);
			/*if (hipGetLastError() != hipSuccess) {
				std::cout << "HopfieldNetGPU_k::updateIS_nodewise, iterazione n: " << this->numIter << std::endl;
				std::cout << hipGetErrorString( hipGetLastError() ) << std::endl; abort();
			}// DEBUG*/
		}

		cuSts = hipMemcpy(&modified, modified_d, sizeof(bool), hipMemcpyDeviceToHost); cudaCheck( cuSts );
		if (this->numIter > ITERATION_LIMIT) {
			std::cout << "Massimo numero di iterazioni raggiunto!!! Uscita forzata" << std::endl;
			break;
		}
	}

	cuSts = hipEventRecord(stop); cudaCheck( cuSts );
	cuSts = hipEventSynchronize(stop); cudaCheck( cuSts );
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
#ifdef VERBOSEHOPFIELD
	std::cout << "Stabilita' raggiunta in " << numIter << " iterazioni" << std::endl;
#endif

	// final state & log
	cuSts = hipMemcpy( this->hState.state, hState_d.state, this->hState.size * sizeof( int ), hipMemcpyDeviceToHost ); cudaCheck( cuSts );
	//HL->GPUrunTime = milliseconds / 1000;
	//HL->GPUnumIter = num_iter;
	//HL->speedup = HL->runTime / HL->GPUrunTime;
	cuSts = hipFree(modified_d); cudaCheck( cuSts );
}



// Kernel alternativo di esecuzione della dinamica della rete
// Ogni nodo e' assegnato ad un thread e calcola sequenzialemente il prodotto
// pesoArco * statoDelVicino accumulandolo in shared memory

/*
	KERNEL FLOW
		1 controllo tid < graphStruct_d->nNodes
		2 controllo colorIdx < col_d->nCol
			se tutto ok creo offsetCol
		3 controllo se (offsetCol + tid) < col_d->cumulSize[colorIdx + 1],
			(se vado a prendere un nodo del colore sbagliato)
		4 shared memory, offset e grado per il nodeIdx
		5 ciclo algoritmo della rete di Hopfield
		6 aggiorno state e score di nodeIdx
		7 controllo se lo stato di nodeIdx è stato modificato
			se sì *modified_d=true
*/

template<typename nodeW, typename edgeW>
__global__ void HopfieldNetGPU_k::updateIS_nodewise( float * const state, float * const score,				// Out values
		const GraphStruct<nodeW, edgeW> * const graphStruct_d,									// graph stuff
		const Coloring * const col_d, const int colorIdx,											// coloring stuff
		bool * const modified_d,																	// stop condition
		const float posState, const float negState ) {												// float const

	unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

	// thread idx supera numero nodi?
	if (tid >= graphStruct_d->nNodes)
		return;

	// color idx supera numero colori?
	if (colorIdx >= col_d->nCol)
		return;

	const int 		offsetCol = col_d->cumulSize[colorIdx];						// offset per il coloring
	//const int 		colSize = col_d->cumulSize[colorIdx + 1] - offsetCol;		// dim del colore attuale

	// ricorda la struttura di colClass e cumulSize nella classe coloring
	// il kernel non deve prendere nodi nodeIdx al di fuori del colore attuale
	if( (offsetCol + tid) >= col_d->cumulSize[colorIdx + 1] )
		return;

	// NOTA per la natura di update_to_standard_notation in ColoringLuby,
	// i nodi di ogni colore sono in ordine crescente di indice
	const int 		nodeIdx = col_d->colClass[offsetCol + tid];					// indice del nodo in cui lavoreremo

	// non alloco prima dei 3 controlli (?)
	extern __shared__ float smem[];
	smem[tid] = 0;

	const int 		offsetDeg	= graphStruct_d->cumulDegs[nodeIdx];		// offset per neighs di nodeIdx
	const int 		degree = graphStruct_d->cumulDegs[nodeIdx+1] - offsetDeg;	// per il ciclo
	unitVal oldState = state[nodeIdx];
	int neighIdx;

	for (int i = 0; i < degree; i++) {
		neighIdx = graphStruct_d->neighs[offsetDeg + i];
		//smem[tid] += (graphStruct_d->nodeWeights[neighIdx] - regulWeight) * state[neighIdx];
		smem[tid] += graphStruct_d->edgeWeights[offsetDeg + i] * state[neighIdx];
	}

	__syncthreads();	// Forse inutile, ma con la shared mem meglio andarci cauti!
/*
	// modifica per la regolarizzazione
	// COMMENTARE
	// la regolazione non viene fatta nell'Hopfiled liscio
	int nodoreg;
	int regOffset;
	int regSize;
	for (int i = 0; i < col_d->nCol; i++) {
		if (i == colorIdx)
			continue;
		else{
			regOffset = col_d->cumulSize[i];
			regSize = col_d->cumulSize[i+1] - regOffset;
			for (int k = 0; k < regSize; k++){
				nodoreg=col_d->colClass[regOffset+k];
				smem[tid] -= state[nodoreg] * regulWeight;
			}
		}
	}

	__syncthreads();*/

	// aggiorno state e score
	score[nodeIdx] = smem[tid] - graphStruct_d->nodeThresholds[nodeIdx];
	state[nodeIdx] = SIGNTH((smem[tid] - graphStruct_d->nodeThresholds[nodeIdx]));

	//controllo se lo stato è stato modificato
	if (state[nodeIdx] != oldState) {
		*modified_d = true;
	}
}



/*
#	#	#	#	#	#	#	#	#	#	#	#	#

		HopfieldNetGPU run & updateIS EDGEWISE

#	#	#	#	#	#	#	#	#	#	#	#	#
*/

//		MAIN MEMBER
//		SINGLE THREAD

/*
	NB ricorda, usa un coloring già in memoria device

	RUN FLOW:
		1 variabili per benchmark, iterazioni e criterio di arresto
		2 copio cumulsize del coloring
		3 CICLO WHILE (finchè gli stati della rete vengono modificati o ITERATION_LIMIT)
			4 CICLO FOR (sui colori/indipendent set)
				lancio dei kernel edgewise per ogni colore

*/

template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::run_edgewise() {
	hipError_t cuSts;

#ifdef PRINTHOPFIELDTITLE
	std::cout << "\033[32;1m** Hopfiled Net GPU runner **\033[0m" << std::endl;
#endif

	//IMPLEMENTATO(?)
	/*
	if (!comprGraph->connected) {
		std::cout << "Warning: graph non connected... EXIT!\n";
		return;
	}*/

	//timer cudaEvent per Benchmark
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//conto iterazioni e criterio di arresto
	unsigned N = graph_d->getStruct()->nNodes;
	this->numIter = 0;
	bool modified = true;
	bool *modified_d;
	cuSts = hipMalloc( (void**) &modified_d, sizeof(bool) ); cudaCheck( cuSts );
	cuSts = hipMemset( modified_d, true, sizeof( bool ) ); cudaCheck( cuSts ); //forse non serve ma andiamo sul sicuro

	//alloco e copio cumulSize
	std::unique_ptr<uint32_t[]> cumulSize_h( new uint32_t[ (col_d->nCol+1) ] );
	cuSts = hipMemcpy( cumulSize_h.get(), col_d->cumulSize, (col_d->nCol+1) * sizeof( uint32_t ), hipMemcpyDeviceToHost ); cudaCheck( cuSts );

#ifdef VERBOSEHOPFIELD
	printf( "Numero colori: %d\n", col->nCol );
	for ( int i = 0; i < col->nCol; i++)
		printf( "colore %d: %d\n", i, ISsize_h[i] );
#endif

	// **  run net on device: loop on ISs  **
	CHECK( hipEventRecord( start ) );
	while ( modified ) {
		/*numIter++;
		CHECK(hipMemset(CUDASTOP_d, 0, sizeof(int)));*/
		this->numIter++;
		cuSts = hipMemset( modified_d, false, sizeof(bool) ); cudaCheck( cuSts );

		// update all ISs
		for (uint32_t ISidx = 0; ISidx < col_d->nCol; ISidx++) {
			// col.meanUnitDeg non implementato nel colorer.
			/*
			int numThreads = pow(2, floor(log(col.meanUnitDeg[ISidx]) / log(2)));
			if (numThreads < 32)
				numThreads = 32;
			if (numThreads > 1024)
				numThreads = 1024;
			*/
			//int numThreads = 32;
			//dim3 blocksize(numThreads);			// num threads = average deg IS nodes crop to (32,1024)
			// col.ISsize[] inaccessibile da host.
			//dim3 gridsize(col.ISsize[ISidx]);	// num blocks = IS size
			//dim3 gridsize( ISsize_h[ISidx] );

			uint32_t colorSize = col_d->cumulSize[ISidx + 1] - col_d->cumulSize[ISidx];
			blocksPerGrid = dim3( colorSize, 1, 1 );

			// launch the Hopfield kernel
			HopfieldNetGPU_k::updateIS_edgewise<<<blocksPerGrid, threadsPerBlock, numThreads * sizeof(float)>>>(
					hState_d.state,			// net state
					hState_d.score,
					graph_d->getStruct(),	// graph structure (neighs, weighs, thresholds)
					col_d,					// graph coloring
					ISidx,					// IS ID
					modified_d,				// stop cond
					this->posState,
					this->negState
					);

			hipDeviceSynchronize();
			if (hipGetLastError() != hipSuccess) { std::cout << "CUDA ERROR: HopfieldNetGPU_k::updateIS_edgewise" << std::endl; abort(); }			// DEBUG

		}

		cuSts = hipMemcpy(&modified, modified_d, sizeof(bool), hipMemcpyDeviceToHost); cudaCheck( cuSts );
		if (this->numIter > ITERATION_LIMIT) {
			std::cout << "Massimo numero di iterazioni raggiunto!!! Uscita forzata" << std::endl;
			break;
		}
	}

	CHECK(hipEventRecord(stop));
	CHECK(hipEventSynchronize(stop));
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
#ifdef VERBOSEHOPFIELD
	std::cout << "Stabilita' raggiunta in " << numIter << " iterazioni" << std::endl;
#endif

	// final state & log
	CHECK(hipMemcpy(this->hState.state, hState_d.state, N * sizeof(int), hipMemcpyDeviceToHost));
	//HL->GPUrunTime = milliseconds / 1000;
	//HL->GPUnumIter = num_iter;
	//HL->speedup = HL->runTime / HL->GPUrunTime;
	cuSts = hipFree(modified_d); cudaCheck( cuSts );
}



// Questo kernel calcola la dinamica della rete di Hopfield.
// Un nodo e' assegnato ad un blocco di thread, fino al massimo numero di blocchi previsti dalla
// configurazione di lancio del kernel.
// Ad ogni thread di un blocco viene assegnato un numero di edge del nodo corrente compatibile
// con il numero massimi di thread prevsti dalla configurazione di lancio, e in esso
// calcola il prodotto pesoArco * statoDelVicino, accumulandolo in shared memory.
// Dopo la fase di calcolo, una reduction additiva calcola il nuovo score e da esso se ne deriva
// il nuovo stato

/*
	KERNEL FLOW
		1 estraggo threadIdx, blockIdx e blockDim
		2 controllo colorIdx < col_d->nCol
			se tutto ok creo offsetCol
		3 controllo se (offsetCol + blockIdx) < col_d->cumulSize[colorIdx + 1],
			(se vado a prendere un nodo del colore sbagliato)
		4 NOTA:
			nodeIdx dipende da blockIdx
			ogni blocco si occupa di un nodo
			ogni thread si occupa di uno o più vicini di nodeIdx
		5 shared memory, offset e grado per il nodeIdx
		6 calcolo interno al thread di quanti edge/vicini deve occuparsi
		7 ciclo algoritmo della rete di Hopfield
			(nella shared mem ogni thread memorizza la sua somma parziale)
		8 somma in parallelo con riduzione, ultimi cicli unrolled
		9 aggiorno state e score di nodeIdx
		10 controllo se lo stato di nodeIdx è stato modificato
			se sì *modified_d=true
			(9 e 10 sono eseguiti dal thread con threadIdx=0 di ogni blocco)
*/

template<typename nodeW, typename edgeW>
__global__ void HopfieldNetGPU_k::updateIS_edgewise( float * const state, float * const score,				// Out values
		const GraphStruct<nodeW, edgeW> * const graphStruct_d,									// graph stuff
		const Coloring * const col_d, const int colorIdx,											// coloring stuff
		bool * const modified_d,																	// stop condition
		const float posState, const float negState ) {

	// ID del thread all'interno del nodo, serve per la parallel reduction sum
	unsigned int tid = threadIdx.x;
	// ID del blocco nella griglia, serve per il calcolo dello score edgewise
	unsigned int bid = blockIdx.x;
	// DIM del blocco 1D, serve per il calcolo dello score edgewise
	// serve per segmentare il calcolo/quanti edge associo ad un thread
	// nel caso il numero di vicini del nodo corrente superi blockDim
	unsigned int dim = blockDim.x;		 // "larghezza" thread:

	// color idx supera numero colori?
	if (colorIdx >= col_d->nCol)
		return;

	const int 		offsetCol = col_d->cumulSize[colorIdx];						// offset per il coloring

	// il kernel non deve prendere nodi nodeIdx al di fuori del colore attuale
	if( (offsetCol + bid) >= col_d->cumulSize[colorIdx + 1] )
		return;

	const int 		nodeIdx = col_d->colClass[offsetCol + bid];					// indice del nodo in cui lavoreremo

	//unsigned int uid = IS[ISidx][bid];   // unit ID
	extern __shared__ float smem[];		 // fissata (per ora) al lancio del kernel come 32 * sizeof(float)

	const int 		offsetDeg	= graphStruct_d->cumulDegs[nodeIdx];				// offset per neighs di nodeIdx
	const int 		degree = graphStruct_d->cumulDegs[nodeIdx+1] - offsetDeg;		// per il ciclo

	// Indica il numero di vicini che ogni thread deve cuccarsi
	// es. se numero vicini = 146 e numero thread per blocco = 32 =>
	// 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 5 4 4 4 4 4 4 4 4 4 4 4 4 4 4
	// es. se numero vicini = 6 e numero thread per blocco = 32 =>
	// 1 1 1 1 1 1 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0 0
	// verifica con:
	// 		int nn = 146; int tPerBlk = 32;
    //		for (int tid = 0; tid < tPerBlk; tid++)
    //		    std::cout << nn / tPerBlk + ((nn % tPerBlk) > tid) << " ";}
	int neighPerThread = degree / blockDim.x + ((degree % blockDim.x) > tid);

	smem[tid] = 0;
	__syncthreads();

	for (int i = 0; i < neighPerThread; i++) {
		int indx = graphStruct_d->neighs[offsetDeg + i * dim + tid];
		smem[tid] += graphStruct_d->edgeWeights[offsetDeg + i * dim + tid] * state[indx];

#ifdef DEBUGPRINTK_IS
		if (bid == 0)
			printf("node: %d tid: %d nperthisThread: %d i: %d idx: %d smem[tid]: %f\n", nodeIdx, tid, neighPerThread, i, indx, smem[tid]);
#endif

	}
	__syncthreads();

	// sum cache by parallel reduction
	for (unsigned int i = dim / 2; i > 32; i >>= 1) {
		if (tid < i)
			smem[tid] += smem[tid + i];
		__syncthreads();
	}
	// last warp
	if (tid < 16) {
		smem[tid] += smem[tid + 16];
		__syncthreads();
		smem[tid] += smem[tid + 8];
		__syncthreads();
		smem[tid] += smem[tid + 4];
		__syncthreads();
		smem[tid] += smem[tid + 2];
		__syncthreads();
		smem[tid] += smem[tid + 1];
		__syncthreads();
	}
	//__syncthreads();

#ifdef DEBUGPRINTK_IS
	// Naive reduction usata in fase di test...
	if (tid == 0) {
		//for( int i = 1; i < dim; i++) {
		//	smem[0] += smem[i];
		//}
		printf("node: %d tid: %d smem[tid]: %f\n", nodeIdx, tid, smem[tid]);
	}
#endif

	// update state
	if (tid == 0) {
		unitVal oldState = state[nodeIdx];
		// aggiorno state e score
		score[nodeIdx] = smem[0] - graphStruct_d->nodeThresholds[nodeIdx];
		state[nodeIdx] = SIGNTH((smem[0] - graphStruct_d->nodeThresholds[nodeIdx]));

		//controllo se lo stato è stato modificato
		if (state[nodeIdx] != oldState) {
			*modified_d = true;
		}
	}
}





/*
#	#	#	#	#	#	#	#	#	#	#	#	#

			HopfieldNetGPU methods

#	#	#	#	#	#	#	#	#	#	#	#	#
*/

// va lasciato hipMemset a 0 per hScore?
template<typename nodeW, typename edgeW>
	void HopfieldNetGPU<nodeW, edgeW>::setInitState( const unitVal * const inState, const  unitVal* const inScore ) {
		hipError_t cuSts;
		for (int i = 0; i < this->hState.size; i++){
			this->hState.state[i] = static_cast<float>(inState[i]);
			//hState.score[i] = static_cast<float>( inScore[i] );
			this->hState.score[i] = 0;
		}
		cuSts = hipMemcpy( hState_d.state, this->hState.state, hState_d.size * sizeof( unitVal ), hipMemcpyHostToDevice ); cudaCheck( cuSts );
		//cuSts = hipMemcpy(hState_d.score, hState.score, hState_d.size * sizeof( unitVal ), hipMemcpyHostToDevice ); cudaCheck( cuSts );
		cuSts = hipMemset( hState_d.score, 0, hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts );
	}

// setta tuti gli initial state = inValue
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::setInitState( const unitVal inValue ) {
	hipError_t cuSts;
	cuSts = hipMemset( hState_d.state, inValue, hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts );
	cuSts = hipMemset( hState_d.score, 0, hState_d.size * sizeof( unitVal ) ); cudaCheck( cuSts );
}

// setta a 0 state e score su memoria device
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::clearInitState() {
	hipError_t cuSts;
	cuSts = hipMemset(hState_d.state, 0, hState_d.size * sizeof( unitVal )); cudaCheck( cuSts );
	cuSts = hipMemset(hState_d.score, 0, hState_d.size * sizeof( unitVal )); cudaCheck( cuSts );
}

// GPURandomizer riempie casualmente state e score su memoria device
// serve ancora?
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::setRandomInitState( GPURand * const randomizer ) {
	//randomizer->fillRandom( hState_d.state, hState_d.size );
}

// vecchia versione?
// serve ancora?
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::setInitStateProb( Prob p, char type ) {
	hipError_t cuSts;
	if ( type == 'z') {
		std::fill( this->hState.state, this->hState.state + this->hState.size, (-0.5 < 0 ? this->negState : this->posState) );
	} else if ( type == 'o') {
		std::fill( this->hState.state, this->hState.state + this->hState.size, (0.5 < 0 ? this->negState : this->posState) );
	} else if ( type == 'r') {
		unitVal pS = this->posState;
		unitVal nS = this->negState;
		std::generate( this->hState.state, this->hState.state + this->hState.size, [p, pS, nS](){return SIGNTHLAMBDA(p-randf(0, 1));} );
	}
	cuSts = hipMemcpy(hState_d.state, this->hState.state, hState_d.size * sizeof( unitVal ), hipMemcpyHostToDevice ); cudaCheck( cuSts );
}

// ritorna i valori di state e score
// serve perchè hState e hState_d sono campi protected
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::returnVal( double * const inState, double * const inScore ) {
	hipError_t cuSts;
	cuSts = hipMemcpy(this->hState.state, hState_d.state, hState_d.size * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts );
	cuSts = hipMemcpy(this->hState.score, hState_d.score, hState_d.size * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts );
	for (int i = 0; i < hState_d.size; i++) {
		inState[i] = static_cast<double>( this->hState.state[i] );
		inScore[i] = static_cast<double>( this->hState.score[i] );
	}
}



/*
#	#	#	#	#	#	#	#	#	#	#	#	#

			HopfieldNetGPU normalize score

#	#	#	#	#	#	#	#	#	#	#	#	#
*/
/*
template<typename nodeW, typename edgeW>
void HopfieldNetGPU<nodeW, edgeW>::normalizeScore( const GraphStruct<nodeW, edgeW> * const bigGraph, const uint32_t *const reduxToFull ) {
	hipError_t cuSts;
	int n = graph_d->getStruct()->nNodes;
	int nOrig = bigGraph->nNodes;
	dim3 threadPerBlk( TPB_ACCUMUL, 1, 1 );
	int bPg = (n + 2 * threadPerBlk.x - 1) / (2 * threadPerBlk.x);
	dim3 blocksPerGrd( bPg, 1, 1 );

	unitVal	*	accumulatedScores;
	int		*	indexes;
	unitVal	*	sumOfWghs;
	cuSts = hipMalloc( (void**)&accumulatedScores, bPg * sizeof( unitVal ) ); cudaCheck( cuSts );
	cuSts = hipMalloc( (void**)&indexes, n * sizeof( int ) ); cudaCheck( cuSts );
	cuSts = hipMalloc( (void**)&sumOfWghs, nOrig * sizeof( unitVal ) ); cudaCheck( cuSts );
	std::unique_ptr<unitVal[]> accumulatedScores_h( new unitVal[bPg] );
	std::unique_ptr<int[]>   indexes_h( new int[n] );
	std::unique_ptr<unitVal[]> sumOfWghs_h( new unitVal[nOrig] );

	//HopfieldNetGPU_k::accumulateDegAndScores <<<blocksPerGrd, threadPerBlk>>> ( n, graph->deg, hState_d.score, accumulatedDeg, accumulatedScores );
	// Calcolo somma degli scores
	HopfieldNetGPUCompr_k::accumulateScores <<<blocksPerGrd, threadPerBlk>>> ( n, hState_d.score, accumulatedScores );
	if (hipGetLastError() != hipSuccess) { std::cout << "CUDA ERROR: HopfieldNetGPUCompr_k::accumulateScores" << std::endl; abort(); }			// DEBUG

	// Calcolo somma dei pesi per ogni nodo del grafo originale
	for (int j = 0; j < nOrig; j++) {
		sumOfWghs_h[j] = std::accumulate(bigGraph->weight[j], bigGraph->weight[j] + bigGraph->deg[j], 0.0);
	}

	int i = 0;
	for_each( indexes_h.get(), indexes_h.get() + n, [reduxToFull, &i]( int &val ) {val = reduxToFull->at( i++ ); });
	// ora indexes_h contiene la mappa reduxToFull per ogni nodo del grafo unlabelled

	// accumulazione della somma dei pesi dei nodi unlabelled
	float accumulatedWDeg = 0.0;
	for (int j = 0; j < n; j++) {
		accumulatedWDeg += sumOfWghs_h[indexes_h[j]];
	}
	hipDeviceSynchronize();

	cuSts = hipMemcpy( accumulatedScores_h.get(), accumulatedScores, bPg * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts );
	cuSts = hipMemcpy( sumOfWghs, sumOfWghs_h.get(), nOrig * sizeof( unitVal ), hipMemcpyHostToDevice ); cudaCheck( cuSts );
	cuSts = hipMemcpy( indexes, indexes_h.get(), n * sizeof( int ), hipMemcpyHostToDevice ); cudaCheck( cuSts );

	// finisco l'accumulazione di deg e score su CPU
	unitVal totScore = std::accumulate( accumulatedScores_h.get(), accumulatedScores_h.get() + bPg, 0.0 );

	bPg = (n + threadPerBlk.x - 1) / threadPerBlk.x;
	blocksPerGrd = dim3( bPg, 1, 1 );
	HopfieldNetGPUCompr_k::normalizeScores <<<blocksPerGrd, threadPerBlk>>> ( n, accumulatedWDeg, totScore, sumOfWghs, indexes, hState_d.score );
	if (hipGetLastError() != hipSuccess) { std::cout << "CUDA ERROR: HopfieldNetGPUCompr_k::normalizeScores" << std::endl; abort(); }			// DEBUG

	cuSts = hipMemcpy( sumOfWghs_h.get(), sumOfWghs, nOrig * sizeof( unitVal ), hipMemcpyDeviceToHost ); cudaCheck( cuSts );
	hipDeviceSynchronize();

	cuSts = hipFree( sumOfWghs ); cudaCheck( cuSts );
	cuSts = hipFree( indexes ); cudaCheck( cuSts );
	cuSts = hipFree( accumulatedScores ); cudaCheck( cuSts );
}
*/

/*
__global__ void HopfieldNetGPU_k::accumulateScores( const int unlab, const unitVal * const scores,
	unitVal * const accumScores ) {

	__shared__ float tempScores[TPB_ACCUMUL];

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= unlab / 2)
		return;
	int baseBlock = 2 * blockDim.x * blockIdx.x;
	tempScores[threadIdx.x] = fabsf(scores[baseBlock + threadIdx.x]);
	__syncthreads();

	int incremento = ((baseBlock + threadIdx.x + blockDim.x) < unlab) ? blockDim.x : (unlab % blockDim.x) / 2;
	tempScores[threadIdx.x] += fabsf(scores[baseBlock + threadIdx.x + incremento]);
	__syncthreads();

	// ahah! Non capisco un cazzo!
#pragma unroll
	for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
		if ((threadIdx.x < i) & (tid + i < unlab / 2)){
			tempScores[threadIdx.x] += tempScores[threadIdx.x + i];
		}
		__syncthreads();
	}

	// Mi prendo cura degli ultimi elementi dei vettori score e deg nel caso in cui
	// i nodi siano dispari
	if ((unlab % 2) && (tid == 0)) {
		tempScores[0] += fabsf(scores[unlab - 1]);
	}
	__syncthreads();

	accumScores[blockIdx.x] = tempScores[0];

	return;
}*/

/*
__global__ void HopfieldNetGPU_k::normalizeScores( const int unlab, const float accumWDeg, const unitVal accumScores,
	const unitVal * const sumOfWeights, const int * const indexes, unitVal * const scores ) {

	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid >= unlab)
		return;

	scores[tid] = sumOfWeights[indexes[tid]] / accumWDeg + scores[tid] / accumScores;

	return;
}
*/


//// Questo serve per mantenere le dechiarazioni e definizioni in classi separate
//// E' necessario aggiungere ogni nuova dichiarazione per ogni nuova classe tipizzata usata nel main
template class HopfieldNetGPU<col, col>;
template class HopfieldNetGPU<float, float>;
